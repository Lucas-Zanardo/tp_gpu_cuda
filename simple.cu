#include "hip/hip_runtime.h"
/**
 * HPC - M2 Data Science - Univ. Lille
 * Authors: C. Bouillaguet and P. Fortin
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define MAX_ITERATIONS 100

#ifndef ELEMENTS_PER_BLOCK
#define ELEMENTS_PER_BLOCK 256
#endif

#include "profile_section.h"
#include "CUDA_common.h"
#include "defs.h"

__global__ void vecMatMultKernel(
        REAL_T *A,
        REAL_T *X,
        REAL_T *Y,
        int n
) {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        int j;
        REAL_T tmp;

        // Split line addition (atomic) -> Occupancy
        tmp = 0;
        for (j = 0; j < n; j++) {
            tmp += A[i * n + j] * X[j];
        }
        Y[i] = tmp;
    }
}

__global__ void normSumKernel(
        REAL_T *Y,
        int n,
        REAL_T *odata
) {
    extern __shared__ REAL_T sdata[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        sdata[tid] = Y[i];
    } else {
        sdata[i] = 0;
    }
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) odata[blockIdx.x] = sdata[0];
}

__global__ void normalizeYKernel(
        REAL_T *Y,
        REAL_T *norm,
        int n
) {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        REAL_T inv_norm = 1.0 / sqrt(*norm);
        Y[i] *= inv_norm;
    }
}

__global__ void errorKernel(
        REAL_T *Y,
        REAL_T *X,
        int n,
        REAL_T *odata
) {
    extern __shared__ REAL_T sdata[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        double delta = X[i] - Y[i];
        sdata[i] = delta * delta;
    } else {
        sdata[i] = 0;
    }
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) odata[blockIdx.x] = sdata[0];
}

__global__ void reduce1(
        REAL_T *indata,
        REAL_T *odata,
        int size
) {
    extern __shared__ REAL_T sdata[];
// each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
        sdata[i] = indata[i];
    else
        sdata[i] = 0;
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) odata[blockIdx.x] = sdata[0];
}



int main(int argc, char **argv) {
    int i, n;
    long long size;
    REAL_T error, norm;
    REAL_T *A, *X, *Y;
    double start_time, total_time;
    int n_iterations;
    FILE *output;

    if (argc < 2) {
        printf("USAGE: %s [n]\n", argv[0]);
        exit(1);
    }
    n = atoi(argv[1]);
    size = (long long) n * n * sizeof(REAL_T);
    printf("Matrix size: %.3f G\n", (double) size / 1073741824.);

    /*** Matrix and vector allocation ***/
    // TODO: Kernel pour initialisation des matrices
    A = (REAL_T *) malloc(size);
    if (A == NULL) {
        perror("Unable to allocate the matrix");
        exit(1);
    }
    X = (REAL_T *) malloc(n * sizeof(REAL_T));
    Y = (REAL_T *) malloc(n * sizeof(REAL_T));
    if ((X == NULL) || (Y == NULL)) {
        perror("Unable to allocate the vectors");
        exit(1);
    }
    /*** Initializing the matrix and x ***/
    for (i = 0; i < n; i++) {
        init_row(A, i, n);
    }

    for (i = 0; i < n; i++) {
        X[i] = 1.0 / n;
    }

    // Kernel dimensions
    dim3 gridSize(ceil((double) n / ELEMENTS_PER_BLOCK));
    dim3 blockSize(ELEMENTS_PER_BLOCK);

    // alloc and transfer data to gpu
    printf("Allocating device memory... ");
    REAL_T *d_A, *d_X, *d_Y;
    REAL_T *d_error, *d_norm;
    REAL_T *d_input_data, *d_output_data;
    hipMalloc((void **) &d_A, size);
    hipMalloc((void **) &d_X, sizeof(REAL_T) * n);
    hipMalloc((void **) &d_Y, sizeof(REAL_T) * n);
    hipMalloc((void **) &d_error, sizeof(REAL_T));
    hipMalloc((void **) &d_norm, sizeof(REAL_T));

    hipMalloc((void **) &d_input_data, sizeof(REAL_T) * gridSize.x);
    hipMalloc((void **) &d_output_data, sizeof(REAL_T) * gridSize.x);
    printf("done\n");
    // transfer data
    printf("Copying data into device... ");
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_X, X, sizeof(REAL_T) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, sizeof(REAL_T) * n, hipMemcpyHostToDevice);
    norm = 0; // reset norm
    printf("done\n");


    start_time = my_gettimeofday();
    {
        n_iterations = 0;
        error = INFINITY;

        printf("GridSize: %d, BlockSize: %d\n", gridSize.x, blockSize.x);
        while (error > ERROR_THRESHOLD && n_iterations < MAX_ITERATIONS) {
            printf("Itération %4d -- Error: %g\n", n_iterations, error);

            {
                profile_cuda_scope("vec mat mult kernel");
                vecMatMultKernel<<<gridSize, blockSize>>>(d_A, d_X, d_Y, n);
            }

            // norm
            {
                profile_cuda_scope("norm sum kernel");
                normSumKernel<<<gridSize, blockSize, sizeof(REAL_T) * blockSize.x>>>(d_Y, n, d_output_data);

                // Reduce
                int num_input = gridSize.x;
                int num_output = gridSize.x / blockSize.x;
                while(num_output > 1) {
                    printf("Error reduce call\n");
                    if (num_input % (blockSize.x))
                        num_output++;
                    reduce1<<<num_output, blockSize.x, sizeof(REAL_T) * blockSize.x>>>(d_input_data, d_output_data, num_input);
                    // swap in and out and reduce one more time if needed
                    num_input = num_output;
                    if (num_output > 1)
                        reduce1<<<num_output, blockSize.x>>>(d_output_data, d_input_data, num_input);
                }
                // copy calculated norm to cpu
                hipMemcpy(&norm, d_output_data, sizeof(REAL_T), hipMemcpyDeviceToHost);
            }

            {
                profile_cuda_scope("normalize y");
                normalizeYKernel<<<gridSize, blockSize>>>(d_Y, &d_output_data[0], n);
            }

            // calculate error
            {
                profile_cuda_scope("error kernel");
                errorKernel<<<gridSize, blockSize, sizeof(REAL_T) * blockSize.x>>>(d_Y, d_X, n, d_output_data);

                // Reduce
                int num_input = gridSize.x;
                int num_output = gridSize.x / blockSize.x;
                while(num_output > 1) {
                    printf("Error reduce call\n");
                    if (num_input % (blockSize.x))
                        num_output++;
                    reduce1<<<num_output, blockSize.x, sizeof(REAL_T) * blockSize.x>>>(d_input_data, d_output_data, num_input);
                    // swap in and out and reduce one more time if needed
                    num_input = num_output;
                    if (num_output > 1)
                        reduce1<<<num_output, blockSize.x>>>(d_output_data, d_input_data, num_input);
                }
                hipMemcpy(&error, d_output_data, sizeof(REAL_T), hipMemcpyDeviceToHost);
                error = sqrt(error);
            }

            // swap device pointers
            REAL_T *d_tmp = d_X;
            d_X = d_Y;
            d_Y = d_tmp;

            n_iterations++;
        }
        // get back eigen vector and norm
        // hipMemcpy(&norm, d_output_data, sizeof(REAL_T), hipMemcpyDeviceToHost);
        norm = sqrt(norm);
        hipMemcpy(X, d_X, sizeof(REAL_T) * n, hipMemcpyDeviceToHost);
    }
    total_time = my_gettimeofday() - start_time;
    printf("final error after %4d iterations: %g (|VP| = %g)\n", n_iterations, error, norm);
    printf("time: %.1f s      Mflop/s: %.1f \n", total_time,
           (2.0 * n * n + 7.0 * n) * n_iterations / 1048576. / total_time);


    /*** Storing the eigen vector in a file ***/
    output = fopen("result.out", "w");
    if (output == NULL) {
        perror("Unable to open result.out in write mode");
        exit(1);
    }
    fprintf(output, "%d\n", n);
    for (i = 0; i < n; i++) {
        fprintf(output, "%.17g\n", X[i]);
    }
    fclose(output);

    free(A);
    hipFree(d_A);
    free(X);
    hipFree(d_X);
    free(Y);
    hipFree(d_Y);
    hipFree(d_input_data);
    hipFree(d_output_data);
}
